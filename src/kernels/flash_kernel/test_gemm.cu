#include"sgemm.cuh"
#include<torch/torch.h>
#include"gemm_cute.cuh"

void test(torch::Tensor a, torch::Tensor b, std::string name){
    if (a.is_cuda())a = a.to(torch::kCPU);
    if (b.is_cuda())b = b.to(torch::kCPU);
    float eps = 1e-1;
    if (a.allclose(b, eps, eps)) {
        std::cout << name << ": pass" << std::endl;
    } else {
        std::cout << name << ": fail" << std::endl;
    }
}

int main(int argc, char* argv[]) {
    if(argc != 4){
        printf("usage: [M] [K] [N]\n");
        exit(0);
    }
    const int M = atoi(argv[1]);
    const int K = atoi(argv[2]);
    const int N = atoi(argv[3]);

    auto A = torch::randn({M, K}, torch::device(torch::kCUDA).dtype(torch::kFloat32));
    auto B = torch::randn({K, N}, torch::device(torch::kCUDA).dtype(torch::kFloat32));
    auto A_half = A.to(torch::kHalf);
    auto B_half = B.to(torch::kHalf);


    auto C1 = sgemm_naive(A, B);
    auto C2 = sgemm_smem(A, B);
    auto C3 = sgemm_smem_reg(A, B);
    auto C4 = sgemm_smem_reg_coalesce(A, B);
    auto C5 = sgemm_smem_reg_coalesce_pg2s(A, B);
    auto C6 = sgemm_smem_reg_coalesce_pg2s_ps2r(A, B);
    auto C7 = cute_sgemm_naive(A, B);
    auto C8 = cute_mma_gemm_simple(A_half, B_half);
    auto C9 = cute_gemm_multi_stage(A_half, B_half.transpose(0, 1).contiguous());
    auto C10 = cute_gemm_multi_stage_demo(A_half, B_half);

    auto C_ref = torch::matmul(A, B);
    auto C_ref_half = torch::matmul(A_half, B_half);

    test(C_ref, C1, "sgemm_naive");
    test(C_ref, C2, "sgemm_smem");
    test(C_ref, C3, "sgemm_smem_reg");
    test(C_ref, C4, "sgemm_smem_reg_coalesce");
    test(C_ref, C5, "sgemm_smem_reg_coalesce_pg2s");
    test(C_ref, C6, "sgemm_smem_reg_coalesce_pg2s_ps2r");
    test(C_ref, C7, "cute_sgemm_naive");
    test(C_ref_half, C8, "cute_mma_gemm_simple");
    test(C_ref_half, C9, "cute_gemm_multi_stage");
    test(C_ref_half, C10, "cute_gemm_multi_stage_demo");

    return 0;
}
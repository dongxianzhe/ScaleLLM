#include <iostream>                                           // 标准输入输出流
#include "cutlass/gemm/device/gemm.h"                         // 引入cutlass头文件
 
using ColumnMajor = cutlass::layout::ColumnMajor;             // 列主序存储方式
using RowMajor    = cutlass::layout::RowMajor;                // 行主序存储方式
 
using CutlassGemm = cutlass::gemm::device::Gemm<float,        // A矩阵数据类型
                                                RowMajor,     // A矩阵存储方式
                                                float,        // B矩阵数据类型
                                                RowMajor,     // B矩阵存储方式
                                                float,        // C矩阵数据类型
                                                RowMajor>;    // C矩阵存储方式
 
void generate_tensor_2D(float *ptr, int i_M, int i_N){        // 二维矩阵填充函数（此处全部填充1）shape (M, N) stride (1, N)
    for(int i = 0; i < i_M; i++){
        for(int j = 0; j < i_N; j++){
            *(ptr + i*i_N + j ) = 1.0;
        }
    }
}
 
int main(int argc, const char *arg[]) {
    int M = 3840;           //M
    int N = 4096;           //N
    int K = 4096;           //K
 
    int lda = K;
    int ldb = K;
    int ldc = N;
    int ldd = N;
 
    float alpha = 1.0;      //alpha
    float beta = 1.0;       //beta
 
    float *A;               //申明A矩阵host端指针
    float *B;               //申明B矩阵host端指针
    float *C;               //申明C矩阵host端指针
    float *D;               //申明D矩阵host端指针
 
    size_t A_mem_size = sizeof(float) * M * K; //memory size of matrix A = M * K * sizeof(float)
    size_t B_mem_size = sizeof(float) * K * N; //memory size of matrix B = K * N * sizeof(float)
    size_t C_mem_size = sizeof(float) * M * N; //memory size of matrix C = M * N * sizeof(float)
    size_t D_mem_size = sizeof(float) * M * N; //memory size of matrix C = M * N * sizeof(float)
 
    A = (float*)malloc(A_mem_size);  // host端A矩阵分配内存
    B = (float*)malloc(B_mem_size);  // host端B矩阵分配内存
    C = (float*)malloc(C_mem_size);  // host端C矩阵分配内存
    D = (float*)malloc(D_mem_size);  // host端D矩阵分配内存
 
    generate_tensor_2D(A, M, K);     // 填充A矩阵
    generate_tensor_2D(B, K, N);     // 填充B矩阵  
    generate_tensor_2D(C, M, N);     // 填充C矩阵  
 
    float *d_A;            // 申明device端A矩阵的指针
    float *d_B;            // 申明device端B矩阵的指针
    float *d_C;            // 申明device端C矩阵的指针
    float *d_D;            // 申明device端D矩阵的指针
 
    hipMalloc((void**)&d_A, A_mem_size);  // device端为A矩阵分配内存
    hipMalloc((void**)&d_B, B_mem_size);  // device端为B矩阵分配内存
    hipMalloc((void**)&d_C, C_mem_size);  // device端为C矩阵分配内存
    hipMalloc((void**)&d_D, D_mem_size);  // device端为D矩阵分配内存
 
    hipMemcpy(d_A, A, A_mem_size, hipMemcpyHostToDevice); // 将矩阵A的数据传递到device端
    hipMemcpy(d_B, B, B_mem_size, hipMemcpyHostToDevice); // 将矩阵B的数据传递到device端
    hipMemcpy(d_C, C, C_mem_size, hipMemcpyHostToDevice); // 将矩阵C的数据传递到device端
 
    CutlassGemm gemm_operator;                  // 申明cutlassgemm类
    CutlassGemm::Arguments args({M, N, K},      // Gemm Problem dimensions
                                {d_A, lda},     // source matrix A
                                {d_B, ldb},     // source matrix B
                                {d_C, ldc},     // source matrix C
                                {d_D, ldd},     // destination matrix D
                                {alpha, beta}); // alpha & beta
    gemm_operator(args); //运行Gemm
 
    hipMemcpy(D, d_D, D_mem_size, hipMemcpyDeviceToHost);  //将运行结果D矩阵传回host端
    std::cout << D[0] << std::endl;                          //打印D中第一行第一个数据
    std::cout << D[M * N - 1] << std::endl;                  //打印D中最后一行最后一个数据
 
    return 0;
}   
 
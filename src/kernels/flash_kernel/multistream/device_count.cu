
#include <hip/hip_runtime.h>
#include<iostream>
#include<vector>

int main(){
    int num_devices;
    hipGetDeviceCount(&num_devices);
    printf("num_devices = %d \n", num_devices);
    std::vector<hipStream_t> streams(num_devices);
    for(int i = 0;i < num_devices;i ++){
        hipSetDevice(i);
        hipStreamCreate(&streams[i]);
    }

    return 0;
}
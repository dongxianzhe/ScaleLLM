#include "hip/hip_runtime.h"
#include<iostream>
#include<torch/torch.h>

template<int block_size>
__global__ void inc_kenrel(float* aptr, float* bptr, int n){
    int i =blockIdx.x * block_size + threadIdx.x;
    if(i < n)bptr[i] = aptr[i] + 1;
}

void inc(torch::Tensor& a, torch::Tensor& b){
    const int n = a.size(0);
    const int block_size = 1024;
    dim3 gridDim((n + block_size - 1) / block_size);
    dim3 blockDim(block_size);
    inc_kenrel<block_size><<<gridDim, block_size, 0, 0>>>(
        static_cast<float*>(a.data_ptr()), 
        static_cast<float*>(b.data_ptr()), 
        n
    );
}

int main(){
    hipStream_t stream;
    hipStreamCreate(&stream);

    int n = 2048;
    auto a = torch::zeros({n}, torch::dtype(torch::kFloat32).device(torch::kCUDA));
    auto b = torch::zeros({n}, torch::dtype(torch::kFloat32).device(torch::kCUDA));

    // performance test
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    bool graphCreated = false;
    hipGraph_t graph;
    hipGraphExec_t instance;
    for(int i = 0;i < 1000;i ++){
        if(!graphCreated){
            hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
            for(int j = 0;j < 20;j ++){
                const int block_size = 1024;
                dim3 gridDim((n + block_size - 1) / block_size);
                dim3 blockDim(block_size);
                inc_kenrel<block_size><<<gridDim, block_size, 0, stream>>>(
                    static_cast<float*>(a.data_ptr()), 
                    static_cast<float*>(b.data_ptr()), 
                    n
                );
            }
            hipStreamEndCapture(stream, &graph);
            hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
            graphCreated = true;
        }
        hipGraphLaunch(instance, stream);
        hipStreamSynchronize(stream);
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    printf("kernel execution %f\n", milliseconds);


    // correctness test
    auto b_ref = a + 1;
    const float eps = 1e-3;
    if(torch::allclose(b_ref, b, eps, eps))puts("passed");
    else puts("failed");

    // puts("---------- b_ref --------------------");
    // std::cout << b_ref.slice(0, 0, 16) << std::endl;
    // puts("---------- b --------------------");
    // std::cout << b.slice(0, 0, 16) << std::endl;

    return 0;
}
#include "hip/hip_runtime.h"
#include<iostream>
#include"simulate_kernels.cuh"
#include<nvml.h>
#include<unistd.h>
#include<hip/hip_runtime.h>

__global__ void cuda_core_kernel(long long num_clocks){
    long long start = clock64();

    volatile int x = 0;
    while((clock64() - start) < num_clocks){
        for(int i = 0;i < 1000000000;i ++){
            x += 1;
        }
    };
}


int main(){
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    std::cout << "multiProcessorCount        : " << deviceProp.multiProcessorCount << std::endl;
    std::cout << "concurrentKernels          : " << deviceProp.concurrentKernels << std::endl;
    std::cout << "clockRate                  : " << deviceProp.clockRate << std::endl;
    std::cout << "maxThreadsPerMultiProcessor: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "sharedMemPerMultiprocessor : " << deviceProp.sharedMemPerMultiprocessor << std::endl;
    std::cout << "regsPerMultiprocessor      : " << deviceProp.regsPerMultiprocessor << std::endl;


    int numBlocks = 0;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, cuda_core_kernel, 769, 0);
    std::cout << "hipOccupancyMaxActiveBlocksPerMultiprocessor: " << numBlocks << std::endl;

    for(int i = 0;i < 1;i ++){
        printf("------------- %d -----------------\n", i);
        hipStream_t stream[3];
        for(int i = 0;i < 3;i ++)hipStreamCreate(&stream[i]);

        hipEvent_t start[3], stop[3];
        for(int i = 0;i < 3;i ++){
            hipEventCreate(&start[i]);
            hipEventCreate(&stop[i]);
        }

        for(int i = 0;i < 3;i ++){
            hipEventRecord(start[i], stream[i]);
        }

        // cuda_core_kernel<<<28, 768, 0, stream[1]>>>(ms2numclock(1000));
        // cuda_core_kernel<<<28, 768, 0, stream[1]>>>(ms2numclock(1000));
        cuda_core_kernel<<<57, 769, 0, stream[2]>>>(ms2numclock(1000));

        hipEventRecord(stop[1], stream[1]);
        hipEventRecord(stop[2], stream[2]);

        hipStreamSynchronize(stream[1]);
        hipStreamSynchronize(stream[2]);
        hipEventRecord(stop[0], stream[0]);

        for(int i = 0;i < 3;i ++){
            hipEventSynchronize(stop[i]);
        }
        // nvmlInit();
        // nvmlDevice_t device;
        // nvmlDeviceGetHandleByIndex(0, &device);
        // nvmlUtilization_t utilization;
        // nvmlDeviceGetUtilizationRates(device, &utilization);
        // nvmlShutdown();
        // printf("utilization %d\n", utilization.gpu);

        for(int i = 0;i < 3;i ++){
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start[i], stop[i]);
            printf("meansure %d %f ms\n", i, milliseconds);
        }

        for(int i = 0;i < 3;i ++){
            hipStreamDestroy(stream[i]);
        }
        sleep(3);
    }
}
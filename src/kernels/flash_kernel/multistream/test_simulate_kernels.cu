#include<iostream>
#include"simulate_kernels.cuh"

void test_busy_wait_kernel(){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    auto time_ms = ms2numclock(1.0);
    busy_wait_kernel<<<1, 128, 0>>>(time_ms);

    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    printf("kernel execution %f ms\n", milliseconds);
}

int main(){
    test_busy_wait_kernel();
}